#include "hip/hip_runtime.h"
#include "forward.h"
#include <hip/hip_runtime.h>

// Logging control - set to 0 to disable all logging, 1 to enable
#define LOG_FORWARD 0

// Error checking macro for CUDA operations
#define CUDA_CHECK(operation, description) \
    cudaStatus = (operation); \
    if (cudaStatus != hipSuccess) { \
        fprintf(stderr, "%s failed: %s\n", (description), hipGetErrorString(cudaStatus)); \
        goto Error; \
    }

// CPU implementations of activation functions
extern "C" {
    double relu(double x) { 
        return x > 0 ? x : 0; 
    }

    double sigmoid(double x) { 
        return 1.0 / (1.0 + exp(-x)); 
    }
}

// CUDA kernel for calculating hidden layer activations with ReLU
__global__ void hidden_layer_kernel(
    const double* input,
    const double* weights,
    const double* bias,
    double* output,
    int input_size,
    int output_size
) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    
    if (i < output_size) {
        double sum = bias[i];
        for (int j = 0; j < input_size; j++) {
            sum += input[j] * weights[j * output_size + i];
        }
        // Apply ReLU activation
        output[i] = sum > 0 ? sum : 0;
    }
}

// CUDA kernel for calculating output layer activations with Sigmoid
__global__ void output_layer_kernel(
    const double* input,
    const double* weights,
    const double* bias,
    double* output,
    int input_size,
    int output_size
) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    
    if (i < output_size) {
        double sum = bias[i];
        for (int j = 0; j < input_size; j++) {
            sum += input[j] * weights[j * output_size + i];
        }
        // Apply Sigmoid activation
        output[i] = 1.0 / (1.0 + exp(-sum));
    }
}

extern "C"
void forward_propagate(
    const double input[INPUT_NODES],
    const double weight1[INPUT_NODES][HIDDEN_NODES],
    const double weight2[HIDDEN_NODES][OUTPUT_NODES],
    const double bias1[HIDDEN_NODES],
    const double bias2[OUTPUT_NODES],
    double hidden[HIDDEN_NODES],
    double output[OUTPUT_NODES],
    int num_threads
) {
    // For logging performance
    static int call_count = 0;
    static double total_time = 0.0;
    static bool first_call = true;
    
    // Only log occasionally to avoid flooding output
    bool should_log = LOG_FORWARD && (call_count % 1000 == 0);
    
    if (LOG_FORWARD && first_call) {
        // Show GPU info on first run
        hipDeviceProp_t deviceProp;
        hipGetDeviceCount(NULL);
        hipGetDeviceProperties(&deviceProp, 0);
        fprintf(stderr, "\n[CUDA] Using GPU: %s\n", deviceProp.name);
        fprintf(stderr, "[CUDA] Compute capability: %d.%d\n", deviceProp.major, deviceProp.minor);
        fprintf(stderr, "[CUDA] Total global memory: %.2f GB\n", deviceProp.totalGlobalMem / (1024.0 * 1024.0 * 1024.0));
        fprintf(stderr, "[CUDA] Multiprocessors: %d\n", deviceProp.multiProcessorCount);
        first_call = false;
    }
    
    // Start timing if logging this call
    clock_t start_time;
    if (should_log) {
        start_time = clock();
        fprintf(stderr, "[CUDA] Forward pass #%d\n", call_count);
    }
    
    // Declare all variables at the top of the function
    hipError_t cudaStatus;
    
    // Device memory pointers - initialize to NULL
    double *d_input = NULL, *d_weight1 = NULL, *d_weight2 = NULL;
    double *d_bias1 = NULL, *d_bias2 = NULL, *d_hidden = NULL, *d_output = NULL;
    
    // Flatten weight matrices
    double weight1_flat[INPUT_NODES * HIDDEN_NODES];
    double weight2_flat[HIDDEN_NODES * OUTPUT_NODES];
    
    // Variables for kernel launch
    int threadsPerBlock = 256;
    int blocksPerGrid;
    
    // Flatten weight matrices for easier CUDA memory handling
    for (int i = 0; i < INPUT_NODES; i++) {
        for (int j = 0; j < HIDDEN_NODES; j++) {
            weight1_flat[i * HIDDEN_NODES + j] = weight1[i][j];
        }
    }
    
    for (int i = 0; i < HIDDEN_NODES; i++) {
        for (int j = 0; j < OUTPUT_NODES; j++) {
            weight2_flat[i * OUTPUT_NODES + j] = weight2[i][j];
        }
    }
    
    if (should_log) {
        fprintf(stderr, "[CUDA] Allocating GPU memory...\n");
    }
    
    // Allocate GPU memory
    CUDA_CHECK(hipMalloc((void**)&d_input, INPUT_NODES * sizeof(double)), "hipMalloc for d_input");
    CUDA_CHECK(hipMalloc((void**)&d_weight1, INPUT_NODES * HIDDEN_NODES * sizeof(double)), "hipMalloc for d_weight1");
    CUDA_CHECK(hipMalloc((void**)&d_bias1, HIDDEN_NODES * sizeof(double)), "hipMalloc for d_bias1");
    CUDA_CHECK(hipMalloc((void**)&d_hidden, HIDDEN_NODES * sizeof(double)), "hipMalloc for d_hidden");
    CUDA_CHECK(hipMalloc((void**)&d_weight2, HIDDEN_NODES * OUTPUT_NODES * sizeof(double)), "hipMalloc for d_weight2");
    CUDA_CHECK(hipMalloc((void**)&d_bias2, OUTPUT_NODES * sizeof(double)), "hipMalloc for d_bias2");
    CUDA_CHECK(hipMalloc((void**)&d_output, OUTPUT_NODES * sizeof(double)), "hipMalloc for d_output");
    
    if (should_log) {
        fprintf(stderr, "[CUDA] Copying data to GPU...\n");
    }
    
    // Copy data from host to device
    CUDA_CHECK(hipMemcpy(d_input, input, INPUT_NODES * sizeof(double), hipMemcpyHostToDevice), "hipMemcpy for d_input");
    CUDA_CHECK(hipMemcpy(d_weight1, weight1_flat, INPUT_NODES * HIDDEN_NODES * sizeof(double), hipMemcpyHostToDevice), "hipMemcpy for d_weight1");
    CUDA_CHECK(hipMemcpy(d_bias1, bias1, HIDDEN_NODES * sizeof(double), hipMemcpyHostToDevice), "hipMemcpy for d_bias1");
    CUDA_CHECK(hipMemcpy(d_weight2, weight2_flat, HIDDEN_NODES * OUTPUT_NODES * sizeof(double), hipMemcpyHostToDevice), "hipMemcpy for d_weight2");
    CUDA_CHECK(hipMemcpy(d_bias2, bias2, OUTPUT_NODES * sizeof(double), hipMemcpyHostToDevice), "hipMemcpy for d_bias2");
    
    if (should_log) {
        fprintf(stderr, "[CUDA] Running hidden layer kernel...\n");
    }
    
    // Launch kernel for hidden layer computation
    blocksPerGrid = (HIDDEN_NODES + threadsPerBlock - 1) / threadsPerBlock;
    
    hidden_layer_kernel<<<blocksPerGrid, threadsPerBlock>>>(
        d_input, d_weight1, d_bias1, d_hidden, INPUT_NODES, HIDDEN_NODES
    );
    
    // Check for kernel launch errors
    CUDA_CHECK(hipGetLastError(), "hidden_layer_kernel launch");
    CUDA_CHECK(hipDeviceSynchronize(), "hipDeviceSynchronize after hidden layer");
    
    if (should_log) {
        fprintf(stderr, "[CUDA] Running output layer kernel...\n");
    }
    
    // Launch kernel for output layer computation
    blocksPerGrid = (OUTPUT_NODES + threadsPerBlock - 1) / threadsPerBlock;
    
    output_layer_kernel<<<blocksPerGrid, threadsPerBlock>>>(
        d_hidden, d_weight2, d_bias2, d_output, HIDDEN_NODES, OUTPUT_NODES
    );
    
    // Check for kernel launch errors
    CUDA_CHECK(hipGetLastError(), "output_layer_kernel launch");
    CUDA_CHECK(hipDeviceSynchronize(), "hipDeviceSynchronize after output layer");
    
    if (should_log) {
        fprintf(stderr, "[CUDA] Copying results back to CPU...\n");
    }
    
    // Copy results back to host
    CUDA_CHECK(hipMemcpy(hidden, d_hidden, HIDDEN_NODES * sizeof(double), hipMemcpyDeviceToHost), "hipMemcpy for hidden");
    CUDA_CHECK(hipMemcpy(output, d_output, OUTPUT_NODES * sizeof(double), hipMemcpyDeviceToHost), "hipMemcpy for output");
    
    // Log performance if needed
    if (should_log) {
        clock_t end_time = clock();
        double elapsed = (double)(end_time - start_time) / CLOCKS_PER_SEC;
        total_time += elapsed;
        
        fprintf(stderr, "[CUDA] Forward pass completed in %.6f seconds\n", elapsed);
        fprintf(stderr, "[CUDA] Average time per pass: %.6f seconds\n", total_time / (call_count + 1));
    }
    
    call_count++;
    
    goto Cleanup;  // Skip error message if successful

Error:
    if (LOG_FORWARD) {
        fprintf(stderr, "[CUDA] Error in forward propagation, call #%d\n", call_count);
    }

Cleanup:
    // Free device memory
    hipFree(d_input);
    hipFree(d_weight1);
    hipFree(d_bias1);
    hipFree(d_hidden);
    hipFree(d_weight2);
    hipFree(d_bias2);
    hipFree(d_output);
} 