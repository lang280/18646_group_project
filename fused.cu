#include "hip/hip_runtime.h"
#include "fused.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>  // Added for hipDeviceptr_t
#include <stdio.h>

// Logging control
#define LOG_FUSED 1

// Error checking macro for CUDA operations
#define CUDA_CHECK(operation, description) \
    do { \
        cudaStatus = (operation); \
        if (cudaStatus != hipSuccess) { \
            fprintf(stderr, "%s failed: %s\n", (description), hipGetErrorString(cudaStatus)); \
            goto Error; \
        } \
    } while(0)

// Modified version for graph creation that doesn't use goto
#define CUDA_CHECK_GRAPH(operation, description, cleanup_action) \
    do { \
        hipError_t localStatus = (operation); \
        if (localStatus != hipSuccess) { \
            fprintf(stderr, "%s failed: %s\n", (description), hipGetErrorString(localStatus)); \
            cleanup_action; \
            return localStatus; \
        } \
    } while(0)

// CUDA thread/block parameters
#define THREADS_PER_BLOCK 256
#define MAX_BLOCKS 65535

// Uncomment to enable CUDA graph optimization
#define USE_CUDA_GRAPH 0

// ────────── Persistent device memory pointers ──────────
// These are allocated once and reused for all batches
static float *d_batch_input = NULL;
static float *d_batch_hidden = NULL; 
static float *d_batch_output = NULL;
static float *d_batch_targets = NULL;
static float *d_batch_hidden_delta = NULL;
static float *d_batch_output_delta = NULL;
static float *d_weight1 = NULL;
static float *d_weight2 = NULL;
static float *d_bias1 = NULL;
static float *d_bias2 = NULL;
static int *d_correct_count = NULL;
static hipStream_t hip_stream = NULL;

// State tracking
static bool is_initialized = false;
static int current_max_batch_size = 0;

// ────────── Activation functions ──────────

// Device functions for activation and derivatives
__device__ inline float relu(float x) {
    return fmaxf(0.0f, x);
}

__device__ inline float relu_derivative(float x) {
    return (x > 0.0f) ? 1.0f : 0.0f;
}

__device__ inline float sigmoid(float x) {
    return 1.0f / (1.0f + expf(-x));
}

__device__ inline float sigmoid_derivative(float y) {
    return y * (1.0f - y);
}

// ────────── Forward pass kernels ──────────

// Optimized kernel for hidden layer forward pass (with ReLU activation)
__global__ void hidden_layer_kernel_batch(
    const float* __restrict__ batch_input,
    const float* __restrict__ weights,
    const float* __restrict__ bias,
    float* __restrict__ batch_hidden,
    int input_size,
    int hidden_size,
    int batch_size
) {
   // Shared memory for bias
    extern __shared__ float shared_mem[];
    float* s_bias = shared_mem;
    
    // Load bias into shared memory (only need first hidden_size elements)
    int tid = threadIdx.x;
    if (tid < hidden_size) {
        s_bias[tid] = bias[tid];
    }
    __syncthreads();
    
    // Calculate global thread ID
    int global_tid = blockIdx.x * blockDim.x + tid;
    
    // Each thread calculates one hidden node activation for one image
    if (global_tid < hidden_size * batch_size) {
        int batch_idx = global_tid / hidden_size;     // Which image in the batch
        int hidden_idx = global_tid % hidden_size;    // Which hidden neuron
        
        // Base pointers for this image/neuron
        const float* input = batch_input + batch_idx * input_size;
        float* hidden = batch_hidden + batch_idx * hidden_size;
        
        // Calculate the sum for this hidden neuron
        float sum = s_bias[hidden_idx];  // Use shared memory for bias
        
        // Calculate dot product using global memory for weights with loop unrolling
        int j = 0;
        
        // Main loop with compiler-directed unrolling
        #pragma unroll 4
        for (j = 0; j < (input_size / 4) * 4; j += 4) {
            sum += input[j] * weights[j * hidden_size + hidden_idx];
            sum += input[j+1] * weights[(j+1) * hidden_size + hidden_idx];
            sum += input[j+2] * weights[(j+2) * hidden_size + hidden_idx];
            sum += input[j+3] * weights[(j+3) * hidden_size + hidden_idx];
        }
        
        // Handle remaining elements
        for (; j < input_size; j++) {
            sum += input[j] * weights[j * hidden_size + hidden_idx];
        }
        
        // Apply ReLU activation and store result
        hidden[hidden_idx] = relu(sum);
    }
}

// Vectorized kernel using float4 for more efficient memory access
__global__ void hidden_layer_kernel_batch_vectorized4(
    const float* __restrict__ batch_input,
    const float* __restrict__ weights,
    const float* __restrict__ bias,
    float* __restrict__ batch_hidden,
    int input_size,
    int hidden_size,
    int batch_size
) {
    // Shared memory for bias
    extern __shared__ float shared_mem[];
    float* s_bias = shared_mem;
    
    // Load bias into shared memory
    int tid = threadIdx.x;
    if (tid < hidden_size) {
        s_bias[tid] = bias[tid];
    }
    __syncthreads();
    
    // Calculate global thread ID
    int global_tid = blockIdx.x * blockDim.x + tid;
    
    // Each thread calculates one hidden node activation for one image
    if (global_tid < hidden_size * batch_size) {
        int batch_idx = global_tid / hidden_size;     // Which image in the batch
        int hidden_idx = global_tid % hidden_size;    // Which hidden neuron
        
        // Base pointers for this image/neuron
        const float* input = batch_input + batch_idx * input_size;
        float* hidden = batch_hidden + batch_idx * hidden_size;
        
        // Cache bias in register
        float sum = s_bias[hidden_idx];
        
        // Cache weight offset in register for better addressing
        int weight_offset = hidden_idx;
        
        // Process four elements at a time using float4 vectorization
        int j = 0;
        int vectorized_limit = (input_size / 4) * 4;  // Round down to multiple of 4
        
        for (; j < vectorized_limit; j += 4) {
            // Load four inputs at once
            float4 input_quad;
            input_quad.x = input[j];
            input_quad.y = input[j+1];
            input_quad.z = input[j+2];
            input_quad.w = input[j+3];
            
            // Load four weights at once
            float4 weight_quad;
            weight_quad.x = weights[j * hidden_size + weight_offset];
            weight_quad.y = weights[(j+1) * hidden_size + weight_offset];
            weight_quad.z = weights[(j+2) * hidden_size + weight_offset];
            weight_quad.w = weights[(j+3) * hidden_size + weight_offset];
            
            // Compute products and accumulate
            sum += input_quad.x * weight_quad.x;
            sum += input_quad.y * weight_quad.y;
            sum += input_quad.z * weight_quad.z;
            sum += input_quad.w * weight_quad.w;
        }
        
        // Handle remaining elements
        for (; j < input_size; j++) {
            sum += input[j] * weights[j * hidden_size + weight_offset];
        }
        
        // Apply ReLU activation and store result
        hidden[hidden_idx] = relu(sum);
    }
}

// Kernel for output layer forward pass (with Sigmoid activation)
__global__ void output_layer_kernel_batch(
    const float* __restrict__ batch_hidden,
    const float* __restrict__ weights,
    const float* __restrict__ bias,
    float* __restrict__ batch_output,
    int hidden_size,
    int output_size,
    int batch_size
) {
    // Shared memory for bias
    extern __shared__ float s_bias[];
    
    // Load bias into shared memory
    int tid = threadIdx.x;
    if (tid < output_size) {
        s_bias[tid] = bias[tid];
    }
    __syncthreads();
    
    // Calculate global thread ID
    int global_tid = blockIdx.x * blockDim.x + tid;
    
    // Each thread calculates one output node activation for one image
    if (global_tid < output_size * batch_size) {
        int batch_idx = global_tid / output_size;     // Which image in the batch
        int output_idx = global_tid % output_size;    // Which output neuron
        
        // Base pointers for this image/neuron
        const float* hidden = batch_hidden + batch_idx * hidden_size;
        float* output = batch_output + batch_idx * output_size;
        
        // Calculate the sum for this output neuron
        float sum = s_bias[output_idx];  // Use shared memory for bias
        
        // Calculate dot product with loop unrolling
        int j = 0;
        
        // Main loop with compiler-directed unrolling
        #pragma unroll 4
        for (j = 0; j < (hidden_size / 4) * 4; j += 4) {
            sum += hidden[j] * weights[j * output_size + output_idx];
            sum += hidden[j+1] * weights[(j+1) * output_size + output_idx];
            sum += hidden[j+2] * weights[(j+2) * output_size + output_idx];
            sum += hidden[j+3] * weights[(j+3) * output_size + output_idx];
        }
        
        // Handle remaining elements
        for (; j < hidden_size; j++) {
            sum += hidden[j] * weights[j * output_size + output_idx];
        }
        
        // Apply Sigmoid activation and store result
        output[output_idx] = sigmoid(sum);
    }
}

// Advanced optimized kernel for output layer forward pass (with Sigmoid activation)
__global__ void output_layer_kernel_batch_advanced(
    const float* __restrict__ batch_hidden,
    const float* __restrict__ weights,
    const float* __restrict__ bias,
    float* __restrict__ batch_output,
    int hidden_size,
    int output_size,
    int batch_size
) {
    // Shared memory for bias
    extern __shared__ float s_bias[];
    
    // Load bias into shared memory
    int tid = threadIdx.x;
    if (tid < output_size) {
        s_bias[tid] = bias[tid];
    }
    __syncthreads();
    
    // Calculate global thread ID
    int global_tid = blockIdx.x * blockDim.x + tid;
    
    // Each thread calculates one output node activation for one image
    if (global_tid < output_size * batch_size) {
        int batch_idx = global_tid / output_size;     // Which image in the batch
        int output_idx = global_tid % output_size;    // Which output neuron
        
        // Base pointers for this image/neuron
        const float* hidden = batch_hidden + batch_idx * hidden_size;
        float* output = batch_output + batch_idx * output_size;
        
        // Cache bias in register
        float sum = s_bias[output_idx];
        
        // Cache weight offset in register for better addressing
        int weight_offset = output_idx;
        
        // Main processing loop with compiler-directed unrolling
        int j = 0;
        #pragma unroll 8
        for (; j < (hidden_size / 8) * 8; j += 8) {
            sum += hidden[j] * weights[j * output_size + weight_offset];
            sum += hidden[j+1] * weights[(j+1) * output_size + weight_offset];
            sum += hidden[j+2] * weights[(j+2) * output_size + weight_offset];
            sum += hidden[j+3] * weights[(j+3) * output_size + weight_offset];
            sum += hidden[j+4] * weights[(j+4) * output_size + weight_offset];
            sum += hidden[j+5] * weights[(j+5) * output_size + weight_offset];
            sum += hidden[j+6] * weights[(j+6) * output_size + weight_offset];
            sum += hidden[j+7] * weights[(j+7) * output_size + weight_offset];
        }
        
        // Handle remaining elements with 4-way unrolling
        #pragma unroll 4
        for (; j < (hidden_size / 4) * 4; j += 4) {
            sum += hidden[j] * weights[j * output_size + weight_offset];
            sum += hidden[j+1] * weights[(j+1) * output_size + weight_offset];
            sum += hidden[j+2] * weights[(j+2) * output_size + weight_offset];
            sum += hidden[j+3] * weights[(j+3) * output_size + weight_offset];
        }
        
        // Handle final elements
        for (; j < hidden_size; j++) {
            sum += hidden[j] * weights[j * output_size + weight_offset];
        }
        
        // Apply Sigmoid activation and store result
        output[output_idx] = sigmoid(sum);
    }
}

// ────────── Backward pass kernels ──────────

// Kernel to calculate output layer deltas
__global__ void calculate_output_delta_kernel(
    const float* __restrict__ batch_output,
    const float* __restrict__ batch_targets,
    float* __restrict__ batch_output_delta,
    int output_size,
    int batch_size
) {
    extern __shared__ float shared_mem[];
    float* s_output = shared_mem;
    float* s_targets = shared_mem + blockDim.x;
    
    int tid = threadIdx.x;
    int global_tid = blockIdx.x * blockDim.x + tid;
    
    // Determine batch and output indices
    if (global_tid < output_size * batch_size) {
        int batch_idx = global_tid / output_size;
        int output_idx = global_tid % output_size;
        
        // Load values into shared memory
        int offset = batch_idx * output_size + output_idx;
        s_output[tid] = batch_output[offset];
        s_targets[tid] = batch_targets[offset];
        
        __syncthreads();
        
        // Compute error and derivative
        float output_val = s_output[tid];
        float target_val = s_targets[tid];
        
        batch_output_delta[offset] = (output_val - target_val) * sigmoid_derivative(output_val);
    }
}

// Kernel to calculate hidden layer deltas
__global__ void calculate_hidden_delta_kernel(
    const float* __restrict__ batch_hidden,
    const float* __restrict__ batch_output_delta,
    const float* __restrict__ weights,
    float* __restrict__ batch_hidden_delta,
    int hidden_size,
    int output_size,
    int batch_size
) {
    extern __shared__ float shared_mem[];
    // Allocate shared memory (weights will be loaded in chunks if needed)
    float* s_hidden = shared_mem;
    float* s_weights_chunk = shared_mem + blockDim.x;
    float* s_output_delta = s_weights_chunk + blockDim.x * output_size;
    
    int tid = threadIdx.x;
    int global_tid = blockIdx.x * blockDim.x + tid;
    
    if (global_tid < hidden_size * batch_size) {
        int batch_idx = global_tid / hidden_size;
        int hidden_idx = global_tid % hidden_size;
        
        // Load hidden value to shared memory
        s_hidden[tid] = batch_hidden[batch_idx * hidden_size + hidden_idx];
        
        // Base pointers for this batch
        const float* output_delta = batch_output_delta + batch_idx * output_size;
        
        // Load output deltas for this batch to shared memory
        if (tid < output_size) {
            s_output_delta[tid] = output_delta[tid];
        }
        
        // Ensure hidden values and output deltas are loaded
        __syncthreads();
        
        // Calculate weighted sum of output deltas
        float sum = 0.0f;
        
        // Process weights in chunks to avoid shared memory limitations
        for (int j = 0; j < output_size; j++) {
            // Load one weight per thread
            if (tid < blockDim.x) {
                s_weights_chunk[tid] = weights[hidden_idx * output_size + j];
            }
            __syncthreads();
            
            sum += s_output_delta[j] * s_weights_chunk[tid % blockDim.x];
        }
        
        // Multiply by derivative of ReLU
        batch_hidden_delta[batch_idx * hidden_size + hidden_idx] = sum * relu_derivative(s_hidden[tid]);
    }
}

// ────────── Weight update kernels ──────────

// Kernel to update weights and biases for hidden layer
__global__ void update_input_hidden_weights_kernel(
    const float* __restrict__ batch_input,
    const float* __restrict__ batch_hidden_delta,
    float* __restrict__ weights,
    float* __restrict__ bias,
    int input_size,
    int hidden_size,
    int batch_size,
    float learning_rate
) {
    extern __shared__ float shared_mem[];
    float* s_gradient_accumulator = shared_mem;
    float* s_bias_gradient = shared_mem + blockDim.x;
    
    int tid = threadIdx.x;
    int global_tid = blockIdx.x * blockDim.x + tid;
    
    // Initialize gradient accumulator in shared memory
    s_gradient_accumulator[tid] = 0.0f;
    
    // Threads handling hidden neuron biases also initialize bias gradient
    if (tid < hidden_size) {
        s_bias_gradient[tid] = 0.0f;
    }
    
    __syncthreads();
    
    if (global_tid < input_size * hidden_size) {
        int input_idx = global_tid / hidden_size;
        int hidden_idx = global_tid % hidden_size;
        
        // Accumulate gradients across batch directly into shared memory
        for (int batch_idx = 0; batch_idx < batch_size; batch_idx++) {
            float input_val = batch_input[batch_idx * input_size + input_idx];
            float hidden_delta = batch_hidden_delta[batch_idx * hidden_size + hidden_idx];
            s_gradient_accumulator[tid] += input_val * hidden_delta;
            
            // For threads handling hidden neurons, also accumulate bias gradients
            if (input_idx == 0 && hidden_idx < hidden_size) {
                s_bias_gradient[hidden_idx] += hidden_delta;
            }
        }
        
        __syncthreads();
        
        // Apply gradient using learning rate
        weights[input_idx * hidden_size + hidden_idx] -= learning_rate * s_gradient_accumulator[tid] / batch_size;
    }
    
    // Update bias for hidden layer (one thread per hidden neuron)
    if (tid < hidden_size && global_tid < input_size * hidden_size) {
        // Apply bias gradient
        bias[tid] -= learning_rate * s_bias_gradient[tid] / batch_size;
    }
}

// Vectorized kernel using float4 for input-hidden weight updates
__global__ void update_input_hidden_weights_kernel_vectorized4(
    const float* __restrict__ batch_input,
    const float* __restrict__ batch_hidden_delta,
    float* __restrict__ weights,
    float* __restrict__ bias,
    int input_size,
    int hidden_size,
    int batch_size,
    float learning_rate
) {
    extern __shared__ float shared_mem[];
    float* s_bias_gradient = shared_mem;
    
    int tid = threadIdx.x;
    int global_tid = blockIdx.x * blockDim.x + tid;
    
    // Initialize gradient accumulator in register for better performance
    float gradient_sum = 0.0f;
    
    // Initialize bias gradients in shared memory
    if (tid < hidden_size) {
        s_bias_gradient[tid] = 0.0f;
    }
    __syncthreads();
    
    if (global_tid < input_size * hidden_size) {
        int input_idx = global_tid / hidden_size;
        int hidden_idx = global_tid % hidden_size;
        
        // Cache indices for better addressing
        int input_offset = input_idx;
        int hidden_offset = hidden_idx;
        
        // Process batches in blocks of 4 for vectorization
        int vectorized_limit = (batch_size / 4) * 4; // Round down to multiple of 4
        
        for (int batch_idx = 0; batch_idx < vectorized_limit; batch_idx += 4) {
            // Load 4 inputs at once
            float4 input_quad;
            input_quad.x = batch_input[(batch_idx+0) * input_size + input_offset];
            input_quad.y = batch_input[(batch_idx+1) * input_size + input_offset];
            input_quad.z = batch_input[(batch_idx+2) * input_size + input_offset];
            input_quad.w = batch_input[(batch_idx+3) * input_size + input_offset];
            
            // Load 4 hidden deltas at once
            float4 delta_quad;
            delta_quad.x = batch_hidden_delta[(batch_idx+0) * hidden_size + hidden_offset];
            delta_quad.y = batch_hidden_delta[(batch_idx+1) * hidden_size + hidden_offset];
            delta_quad.z = batch_hidden_delta[(batch_idx+2) * hidden_size + hidden_offset];
            delta_quad.w = batch_hidden_delta[(batch_idx+3) * hidden_size + hidden_offset];
            
            // Compute gradient contributions and accumulate
            gradient_sum += input_quad.x * delta_quad.x;
            gradient_sum += input_quad.y * delta_quad.y;
            gradient_sum += input_quad.z * delta_quad.z;
            gradient_sum += input_quad.w * delta_quad.w;
            
            // Accumulate bias gradients if this thread handles a bias
            if (input_idx == 0) {
                atomicAdd(&s_bias_gradient[hidden_idx], delta_quad.x + delta_quad.y + delta_quad.z + delta_quad.w);
            }
        }
        
        // Handle the remaining items (less than 4)
        for (int batch_idx = vectorized_limit; batch_idx < batch_size; batch_idx++) {
            float input_val = batch_input[batch_idx * input_size + input_offset];
            float hidden_delta = batch_hidden_delta[batch_idx * hidden_size + hidden_offset];
            
            gradient_sum += input_val * hidden_delta;
            
            if (input_idx == 0) {
                atomicAdd(&s_bias_gradient[hidden_idx], hidden_delta);
            }
        }
        
        __syncthreads();
        
        // Update weight with a single write operation
        weights[input_idx * hidden_size + hidden_idx] -= learning_rate * gradient_sum / batch_size;
        
        // Update bias if this thread is responsible
        if (input_idx == 0 && hidden_idx < hidden_size) {
            bias[hidden_idx] -= learning_rate * s_bias_gradient[hidden_idx] / batch_size;
        }
    }
}

// Kernel to update weights and biases for output layer
__global__ void update_hidden_output_weights_kernel(
    const float* __restrict__ batch_hidden,
    const float* __restrict__ batch_output_delta,
    float* __restrict__ weights,
    float* __restrict__ bias,
    int hidden_size,
    int output_size,
    int batch_size,
    float learning_rate
) {
    extern __shared__ float shared_mem[];
    float* s_gradient_accumulator = shared_mem;
    float* s_bias_gradient = shared_mem + blockDim.x;
    
    int tid = threadIdx.x;
    int global_tid = blockIdx.x * blockDim.x + tid;
    
    // Initialize gradient accumulators in shared memory
    s_gradient_accumulator[tid] = 0.0f;
    
    // Threads handling output neuron biases also initialize bias gradient
    if (tid < output_size) {
        s_bias_gradient[tid] = 0.0f;
    }
    
    __syncthreads();
    
    if (global_tid < hidden_size * output_size) {
        int hidden_idx = global_tid / output_size;
        int output_idx = global_tid % output_size;
        
        // Accumulate gradients across batch into shared memory
        for (int batch_idx = 0; batch_idx < batch_size; batch_idx++) {
            float hidden_val = batch_hidden[batch_idx * hidden_size + hidden_idx];
            float output_delta = batch_output_delta[batch_idx * output_size + output_idx];
            s_gradient_accumulator[tid] += hidden_val * output_delta;
            
            // For threads handling output neurons, also accumulate bias gradients
            if (hidden_idx == 0 && output_idx < output_size) {
                s_bias_gradient[output_idx] += output_delta;
            }
        }
        
        __syncthreads();
        
        // Apply gradient using learning rate
        weights[hidden_idx * output_size + output_idx] -= learning_rate * s_gradient_accumulator[tid] / batch_size;
    }
    
    // Update bias for output layer (one thread per output neuron)
    if (tid < output_size && global_tid < hidden_size * output_size) {
        // Apply bias gradient
        bias[tid] -= learning_rate * s_bias_gradient[tid] / batch_size;
    }
}

// Kernel to count correct predictions in the batch
__global__ void count_correct_predictions_kernel(
    const float* __restrict__ batch_output,
    const float* __restrict__ batch_targets,
    int* __restrict__ correct_count,
    int output_size,
    int batch_size
) {
    // Shared memory for local counts (one per thread block)
    __shared__ int block_correct_count;
    
    // Initialize shared memory
    if (threadIdx.x == 0) {
        block_correct_count = 0;
    }
    __syncthreads();
    
    // Process one image per thread
    int batch_idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (batch_idx < batch_size) {
        // Find the predicted class (max output value)
        int predicted_class = 0;
        float max_output = batch_output[batch_idx * output_size];
        
        for (int j = 1; j < output_size; j++) {
            float output_val = batch_output[batch_idx * output_size + j];
            if (output_val > max_output) {
                max_output = output_val;
                predicted_class = j;
            }
        }
        
        // Find the target class (index of 1.0)
        int target_class = 0;
        for (int j = 0; j < output_size; j++) {
            if (batch_targets[batch_idx * output_size + j] > 0.5f) {
                target_class = j;
                break;
            }
        }
        
        // Increment the count if prediction matches target
        if (predicted_class == target_class) {
            atomicAdd(&block_correct_count, 1);
        }
    }
    
    // Wait for all threads to finish
    __syncthreads();
    
    // Add block count to global count
    if (threadIdx.x == 0) {
        atomicAdd(correct_count, block_correct_count);
    }
}

// ────────── CUDA Graph structures and setup ──────────
#ifdef USE_CUDA_GRAPH
typedef struct {
    // Graph objects
    hipGraph_t graph;
    hipGraphExec_t instance;
    
    // Whether graph has been created
    bool initialized;
    
    // Batch size used for graph creation
    int captured_batch_size;
    float captured_learning_rate;
    
    // Parameters needed for operation
    int input_size;
    int hidden_size;
    int output_size;
} GraphData;

// Global graph data
static GraphData graph_data = {0};

// Function to clean up graph resources
void cleanup_graph_resources() {
    if (graph_data.initialized) {
        if (graph_data.instance) {
            hipGraphExecDestroy(graph_data.instance);
            graph_data.instance = NULL;
        }
        if (graph_data.graph) {
            hipGraphDestroy(graph_data.graph);
            graph_data.graph = NULL;
        }
        graph_data.initialized = false;
    }
}

// Function to clean up CUDA resources
void cleanup_cuda_resources(hipGraph_t graph, hipGraphExec_t instance) {
    if (instance) {
        hipGraphExecDestroy(instance);
    }
    if (graph) {
        hipGraphDestroy(graph);
    }
}

// Function to create and initialize a CUDA graph for training
hipError_t create_training_graph(
    int batch_size,
    float learning_rate
) {
    // Initialize all variables at the beginning
    hipGraph_t tempGraph = NULL;
    hipGraphExec_t tempInstance = NULL;
    hipGraphNode_t errorNode = NULL;

    // Calculate grid dimensions for kernels
    int total_hidden_neurons = batch_size * HIDDEN_NODES;
    int hidden_blocks = (total_hidden_neurons + THREADS_PER_BLOCK - 1) / THREADS_PER_BLOCK;
    hidden_blocks = hidden_blocks > MAX_BLOCKS ? MAX_BLOCKS : hidden_blocks;
    
    int total_output_neurons = batch_size * OUTPUT_NODES;
    int output_blocks = (total_output_neurons + THREADS_PER_BLOCK - 1) / THREADS_PER_BLOCK;
    output_blocks = output_blocks > MAX_BLOCKS ? MAX_BLOCKS : output_blocks;
    
    int prediction_blocks = (batch_size + THREADS_PER_BLOCK - 1) / THREADS_PER_BLOCK;
    
    int input_hidden_blocks = (INPUT_NODES * HIDDEN_NODES + THREADS_PER_BLOCK - 1) / THREADS_PER_BLOCK;
    input_hidden_blocks = input_hidden_blocks > MAX_BLOCKS ? MAX_BLOCKS : input_hidden_blocks;
    
    int hidden_output_blocks = (HIDDEN_NODES * OUTPUT_NODES + THREADS_PER_BLOCK - 1) / THREADS_PER_BLOCK;
    hidden_output_blocks = hidden_output_blocks > MAX_BLOCKS ? MAX_BLOCKS : hidden_output_blocks;

    // Use non-goto error checking for graph creation
    hipError_t status;
    
    // Begin capturing
    status = hipStreamBeginCapture(hip_stream, hipStreamCaptureModeGlobal);
    if (status != hipSuccess) {
        fprintf(stderr, "Begin stream capture failed: %s\n", hipGetErrorString(status));
        return status;
    }
    
    // Reset correct count
    status = hipMemsetAsync(d_correct_count, 0, sizeof(int), hip_stream);
    if (status != hipSuccess) {
        fprintf(stderr, "Reset correct count failed: %s\n", hipGetErrorString(status));
        status = hipStreamEndCapture(hip_stream, &tempGraph);
        return hipErrorUnknown;
    }
    
    // Forward pass - hidden layer (using vectorized4 version)
    hidden_layer_kernel_batch_vectorized4<<<hidden_blocks, THREADS_PER_BLOCK, HIDDEN_NODES * sizeof(float), hip_stream>>>(
        d_batch_input, d_weight1, d_bias1, d_batch_hidden, 
        INPUT_NODES, HIDDEN_NODES, batch_size
    );
    
    // Forward pass - output layer
    output_layer_kernel_batch<<<output_blocks, THREADS_PER_BLOCK, OUTPUT_NODES * sizeof(float), hip_stream>>>(
        d_batch_hidden, d_weight2, d_bias2, d_batch_output, 
        HIDDEN_NODES, OUTPUT_NODES, batch_size
    );
    
    // Count correct predictions
    count_correct_predictions_kernel<<<prediction_blocks, THREADS_PER_BLOCK, 0, hip_stream>>>(
        d_batch_output, d_batch_targets, d_correct_count, 
        OUTPUT_NODES, batch_size
    );
    
    // Backward pass - output layer deltas
    calculate_output_delta_kernel<<<output_blocks, THREADS_PER_BLOCK, 2 * THREADS_PER_BLOCK * sizeof(float), hip_stream>>>(
        d_batch_output, d_batch_targets, d_batch_output_delta, 
        OUTPUT_NODES, batch_size
    );
    
    // Backward pass - hidden layer deltas
    calculate_hidden_delta_kernel<<<hidden_blocks, THREADS_PER_BLOCK, 
        (THREADS_PER_BLOCK + THREADS_PER_BLOCK * OUTPUT_NODES + OUTPUT_NODES) * sizeof(float), hip_stream>>>(
        d_batch_hidden, d_batch_output_delta, d_weight2, d_batch_hidden_delta, 
        HIDDEN_NODES, OUTPUT_NODES, batch_size
    );
    
    // Weight updates - input to hidden (using vectorized4 version)
    update_input_hidden_weights_kernel_vectorized4<<<input_hidden_blocks, THREADS_PER_BLOCK, 
        HIDDEN_NODES * sizeof(float), hip_stream>>>(
        d_batch_input, d_batch_hidden_delta, d_weight1, d_bias1,
        INPUT_NODES, HIDDEN_NODES, batch_size, learning_rate
    );
    
    // Weight updates - hidden to output
    update_hidden_output_weights_kernel<<<hidden_output_blocks, THREADS_PER_BLOCK, 
        (THREADS_PER_BLOCK + OUTPUT_NODES) * sizeof(float), hip_stream>>>(
        d_batch_hidden, d_batch_output_delta, d_weight2, d_bias2,
        HIDDEN_NODES, OUTPUT_NODES, batch_size, learning_rate
    );
    
    // End capturing
    status = hipStreamEndCapture(hip_stream, &tempGraph);
    if (status != hipSuccess) {
        fprintf(stderr, "End stream capture failed: %s\n", hipGetErrorString(status));
        return status;
    }
    
    // Create executable graph
    status = hipGraphInstantiate(&tempInstance, tempGraph, &errorNode, NULL, 0);
    if (status != hipSuccess) {
        fprintf(stderr, "Instantiate graph failed: %s\n", hipGetErrorString(status));
        cleanup_cuda_resources(tempGraph, NULL);
        return status;
    }
    
    // Cleanup any existing graph
    cleanup_graph_resources();
    
    // Store the new graph
    graph_data.graph = tempGraph;
    graph_data.instance = tempInstance;
    graph_data.captured_batch_size = batch_size;
    graph_data.captured_learning_rate = learning_rate;
    graph_data.input_size = INPUT_NODES;
    graph_data.hidden_size = HIDDEN_NODES;
    graph_data.output_size = OUTPUT_NODES;
    
    // Mark graph as initialized
    graph_data.initialized = true;
    
    return hipSuccess;
}
#endif

// ────────── Initialization and cleanup functions ──────────

// Initialize the CUDA training environment
extern "C"
hipError_t init_fused_training(int max_batch_size) {
    if (is_initialized && max_batch_size <= current_max_batch_size) {
        // Already initialized with sufficient capacity
        return hipSuccess;
    }
    
    // Cleanup previous resources if any
    cleanup_fused_training();
    
    hipError_t cudaStatus;
    
    // Create stream
    // CUDA_CHECK(hipStreamCreate(&hip_stream), "Create CUDA stream");
    hipStreamCreate(&hip_stream);
    
    // Allocate device memory with capacity for the maximum batch size
    size_t batch_input_size = max_batch_size * INPUT_NODES * sizeof(float);
    size_t batch_hidden_size = max_batch_size * HIDDEN_NODES * sizeof(float);
    size_t batch_output_size = max_batch_size * OUTPUT_NODES * sizeof(float);
    size_t weight1_size = INPUT_NODES * HIDDEN_NODES * sizeof(float);
    size_t weight2_size = HIDDEN_NODES * OUTPUT_NODES * sizeof(float);
    size_t bias1_size = HIDDEN_NODES * sizeof(float);
    size_t bias2_size = OUTPUT_NODES * sizeof(float);
    
    // Allocate memory for inputs, activations, deltas
    CUDA_CHECK(hipMalloc((void**)&d_batch_input, batch_input_size), "hipMalloc for batch input");
    CUDA_CHECK(hipMalloc((void**)&d_batch_hidden, batch_hidden_size), "hipMalloc for batch hidden");
    CUDA_CHECK(hipMalloc((void**)&d_batch_output, batch_output_size), "hipMalloc for batch output");
    CUDA_CHECK(hipMalloc((void**)&d_batch_targets, batch_output_size), "hipMalloc for batch targets");
    CUDA_CHECK(hipMalloc((void**)&d_batch_hidden_delta, batch_hidden_size), "hipMalloc for hidden delta");
    CUDA_CHECK(hipMalloc((void**)&d_batch_output_delta, batch_output_size), "hipMalloc for output delta");
    
    // Allocate memory for weights and biases
    CUDA_CHECK(hipMalloc((void**)&d_weight1, weight1_size), "hipMalloc for weight1");
    CUDA_CHECK(hipMalloc((void**)&d_weight2, weight2_size), "hipMalloc for weight2");
    CUDA_CHECK(hipMalloc((void**)&d_bias1, bias1_size), "hipMalloc for bias1");
    CUDA_CHECK(hipMalloc((void**)&d_bias2, bias2_size), "hipMalloc for bias2");
    
    // Allocate memory for correct prediction count
    CUDA_CHECK(hipMalloc((void**)&d_correct_count, sizeof(int)), "hipMalloc for correct count");
    
    // Update state
    is_initialized = true;
    current_max_batch_size = max_batch_size;
    
    return hipSuccess;
    
Error:
    // Cleanup on error
    cleanup_fused_training();
    return cudaStatus;
}

// Cleanup all CUDA resources
extern "C"
void cleanup_fused_training() {
    // Clean up graph
    #ifdef USE_CUDA_GRAPH
    cleanup_graph_resources();
    #endif
    
    // Free device memory
    if (d_batch_input) hipFree(d_batch_input);
    if (d_batch_hidden) hipFree(d_batch_hidden);
    if (d_batch_output) hipFree(d_batch_output);
    if (d_batch_targets) hipFree(d_batch_targets);
    if (d_batch_hidden_delta) hipFree(d_batch_hidden_delta);
    if (d_batch_output_delta) hipFree(d_batch_output_delta);
    if (d_weight1) hipFree(d_weight1);
    if (d_weight2) hipFree(d_weight2);
    if (d_bias1) hipFree(d_bias1);
    if (d_bias2) hipFree(d_bias2);
    if (d_correct_count) hipFree(d_correct_count);
    
    // Destroy stream
    if (hip_stream) hipStreamDestroy(hip_stream);
    
    // Reset pointers
    d_batch_input = NULL;
    d_batch_hidden = NULL;
    d_batch_output = NULL;
    d_batch_targets = NULL;
    d_batch_hidden_delta = NULL;
    d_batch_output_delta = NULL;
    d_weight1 = NULL;
    d_weight2 = NULL;
    d_bias1 = NULL;
    d_bias2 = NULL;
    d_correct_count = NULL;
    hip_stream = NULL;
    
    // Reset state
    is_initialized = false;
    current_max_batch_size = 0;
}

// ────────── Main fused training function ──────────

extern "C"
void train_batch_fused(
    const float* batch_input,
    const float* batch_targets,
    float* weight1,
    float* weight2, 
    float* bias1,
    float* bias2,
    int batch_size,
    float learning_rate,
    int* correct_predictions
)
{
    // For performance logging
    static int call_count = 0;
    static double total_time = 0.0;
    clock_t start_time = 0, end_time = 0;
    
    // Only log occasionally
    bool should_log = LOG_FUSED && (call_count % 10 == 0);
    
    if (should_log) {
        start_time = clock();
        printf("[FUSED] Starting batch #%d with %d images\n", call_count, batch_size);
    }
    
    // CUDA error status
    hipError_t cudaStatus;
    
    // Host-side counter for correct predictions
    int host_correct_count = 0;
    
    // Flag to indicate if we're using CUDA graph
    bool using_graph = false;
    
    // Ensure environment is initialized
    if (!is_initialized || batch_size > current_max_batch_size) {
        if (should_log) {
            printf("[FUSED] Initializing training environment with batch size %d\n", batch_size);
        }
        // CUDA_CHECK(init_fused_training(batch_size), "Initialize fused training");
        init_fused_training(batch_size);
    }
    
    // Reset correct count
    // CUDA_CHECK(hipMemset(d_correct_count, 0, sizeof(int)), "Reset correct count");
    hipMemset(d_correct_count, 0, sizeof(int));
    
    if (should_log) {
        printf("[FUSED] Copying data to device...\n");
    }
    
    // Copy input data, weights, and biases to device
    size_t batch_input_size = batch_size * INPUT_NODES * sizeof(float);
    size_t batch_output_size = batch_size * OUTPUT_NODES * sizeof(float);
    size_t weight1_size = INPUT_NODES * HIDDEN_NODES * sizeof(float);
    size_t weight2_size = HIDDEN_NODES * OUTPUT_NODES * sizeof(float);
    size_t bias1_size = HIDDEN_NODES * sizeof(float);
    size_t bias2_size = OUTPUT_NODES * sizeof(float);
    
    CUDA_CHECK(hipMemcpy(d_batch_input, batch_input, batch_input_size, hipMemcpyHostToDevice), "hipMemcpy for batch input");
    CUDA_CHECK(hipMemcpy(d_batch_targets, batch_targets, batch_output_size, hipMemcpyHostToDevice), "hipMemcpy for batch targets");
    CUDA_CHECK(hipMemcpy(d_weight1, weight1, weight1_size, hipMemcpyHostToDevice), "hipMemcpy for weight1");
    CUDA_CHECK(hipMemcpy(d_weight2, weight2, weight2_size, hipMemcpyHostToDevice), "hipMemcpy for weight2");
    CUDA_CHECK(hipMemcpy(d_bias1, bias1, bias1_size, hipMemcpyHostToDevice), "hipMemcpy for bias1");
    CUDA_CHECK(hipMemcpy(d_bias2, bias2, bias2_size, hipMemcpyHostToDevice), "hipMemcpy for bias2");
    
#ifdef USE_CUDA_GRAPH
    // Check if we can use CUDA graph
    if (graph_data.initialized && 
        batch_size == graph_data.captured_batch_size && 
        learning_rate == graph_data.captured_learning_rate) {
        
        // We can reuse the existing graph
        if (should_log) {
            printf("[FUSED] Executing existing CUDA graph...\n");
        }
        
        CUDA_CHECK(hipGraphLaunch(graph_data.instance, hip_stream), "Launch graph");
        CUDA_CHECK(hipStreamSynchronize(hip_stream), "Stream synchronize after graph launch");
        
        using_graph = true;
    } else {
        // Need to create a new graph for this batch size/learning rate
        if (should_log) {
            printf("[FUSED] Creating new CUDA graph for batch size %d and learning rate %.5f\n", 
                  batch_size, learning_rate);
        }
        
        // Create a new graph
        cudaStatus = create_training_graph(batch_size, learning_rate);
            
        if (cudaStatus == hipSuccess) {
            // Execute the graph
            CUDA_CHECK(hipGraphLaunch(graph_data.instance, hip_stream), "Launch graph");
            CUDA_CHECK(hipStreamSynchronize(hip_stream), "Stream synchronize after graph launch");
            
            using_graph = true;
        } else {
            // Fall back to regular execution
            cleanup_graph_resources();
            if (should_log) {
                printf("[FUSED] Falling back to regular execution...\n");
            }
        }
    }
#endif

    // If not using graph, execute operations normally
    if (!using_graph) {
        // Create CUDA events for kernel timing
        hipEvent_t kernel_start, kernel_stop;
        float kernel_time;
        CUDA_CHECK(hipEventCreate(&kernel_start), "Create kernel start event");
        CUDA_CHECK(hipEventCreate(&kernel_stop), "Create kernel stop event");
        
        // ────────── Forward Pass ──────────
        if (should_log) {
            printf("[FUSED] Performing forward pass...\n");
        }
        
        // Calculate grid dimensions for hidden layer
        int total_hidden_neurons = batch_size * HIDDEN_NODES;
        int hidden_blocks = (total_hidden_neurons + THREADS_PER_BLOCK - 1) / THREADS_PER_BLOCK;
        hidden_blocks = hidden_blocks > MAX_BLOCKS ? MAX_BLOCKS : hidden_blocks;
        
        // Time hidden layer kernel
        CUDA_CHECK(hipEventRecord(kernel_start, hip_stream), "Record hidden layer start event");
        
        // Launch hidden layer kernel (using vectorized4 version)
        hidden_layer_kernel_batch_vectorized4<<<hidden_blocks, THREADS_PER_BLOCK, HIDDEN_NODES * sizeof(float), hip_stream>>>(
            d_batch_input, d_weight1, d_bias1, d_batch_hidden, 
            INPUT_NODES, HIDDEN_NODES, batch_size
        );
        
        CUDA_CHECK(hipEventRecord(kernel_stop, hip_stream), "Record hidden layer stop event");
        CUDA_CHECK(hipEventSynchronize(kernel_stop), "Synchronize hidden layer stop event");
        CUDA_CHECK(hipEventElapsedTime(&kernel_time, kernel_start, kernel_stop), "Calculate hidden layer time");
        if (should_log) printf("[TIMING] Hidden layer kernel: %.4f ms\n", kernel_time);
        
        // Check for kernel launch errors
        CUDA_CHECK(hipGetLastError(), "hidden_layer_kernel_batch_vectorized4 launch");
        
        // Calculate grid dimensions for output layer
        int total_output_neurons = batch_size * OUTPUT_NODES;
        int output_blocks = (total_output_neurons + THREADS_PER_BLOCK - 1) / THREADS_PER_BLOCK;
        output_blocks = output_blocks > MAX_BLOCKS ? MAX_BLOCKS : output_blocks;
        
        // Time output layer kernel
        CUDA_CHECK(hipEventRecord(kernel_start, hip_stream), "Record output layer start event");
        
        // Launch output layer kernel
        output_layer_kernel_batch<<<output_blocks, THREADS_PER_BLOCK, OUTPUT_NODES * sizeof(float), hip_stream>>>(
            d_batch_hidden, d_weight2, d_bias2, d_batch_output, 
            HIDDEN_NODES, OUTPUT_NODES, batch_size
        );
        
        CUDA_CHECK(hipEventRecord(kernel_stop, hip_stream), "Record output layer stop event");
        CUDA_CHECK(hipEventSynchronize(kernel_stop), "Synchronize output layer stop event");
        CUDA_CHECK(hipEventElapsedTime(&kernel_time, kernel_start, kernel_stop), "Calculate output layer time");
        if (should_log) printf("[TIMING] Output layer kernel: %.4f ms\n", kernel_time);
        
        // Check for kernel launch errors
        CUDA_CHECK(hipGetLastError(), "output_layer_kernel_batch launch");
        
        // ────────── Count Correct Predictions ──────────
        if (should_log) {
            printf("[FUSED] Counting correct predictions...\n");
        }
        
        // Time prediction kernel
        CUDA_CHECK(hipEventRecord(kernel_start, hip_stream), "Record prediction start event");
        
        // Launch kernel to count correct predictions
        int prediction_blocks = (batch_size + THREADS_PER_BLOCK - 1) / THREADS_PER_BLOCK;
        count_correct_predictions_kernel<<<prediction_blocks, THREADS_PER_BLOCK, 0, hip_stream>>>(
            d_batch_output, d_batch_targets, d_correct_count, 
            OUTPUT_NODES, batch_size
        );
        
        CUDA_CHECK(hipEventRecord(kernel_stop, hip_stream), "Record prediction stop event");
        CUDA_CHECK(hipEventSynchronize(kernel_stop), "Synchronize prediction stop event");
        CUDA_CHECK(hipEventElapsedTime(&kernel_time, kernel_start, kernel_stop), "Calculate prediction time");
        if (should_log) printf("[TIMING] Count predictions kernel: %.4f ms\n", kernel_time);
        
        // Check for kernel launch errors
        CUDA_CHECK(hipGetLastError(), "count_correct_predictions_kernel launch");
        
        // ────────── Backward Pass ──────────
        if (should_log && learning_rate > 0.0f) {
            printf("[FUSED] Performing backward pass...\n");
        }
        
        // Only perform backward pass if we're training (learning_rate > 0)
        if (learning_rate > 0.0f) {
            // Time output delta kernel
            CUDA_CHECK(hipEventRecord(kernel_start, hip_stream), "Record output delta start event");
            
            // Calculate output layer deltas
            calculate_output_delta_kernel<<<output_blocks, THREADS_PER_BLOCK, 2 * THREADS_PER_BLOCK * sizeof(float), hip_stream>>>(
                d_batch_output, d_batch_targets, d_batch_output_delta, 
                OUTPUT_NODES, batch_size
            );
            
            CUDA_CHECK(hipEventRecord(kernel_stop, hip_stream), "Record output delta stop event");
            CUDA_CHECK(hipEventSynchronize(kernel_stop), "Synchronize output delta stop event");
            CUDA_CHECK(hipEventElapsedTime(&kernel_time, kernel_start, kernel_stop), "Calculate output delta time");
            if (should_log) printf("[TIMING] Output delta kernel: %.4f ms\n", kernel_time);
            
            // Check for kernel launch errors
            CUDA_CHECK(hipGetLastError(), "calculate_output_delta_kernel launch");
            
            // Time hidden delta kernel
            CUDA_CHECK(hipEventRecord(kernel_start, hip_stream), "Record hidden delta start event");
            
            // Calculate hidden layer deltas
            calculate_hidden_delta_kernel<<<hidden_blocks, THREADS_PER_BLOCK, 
                (THREADS_PER_BLOCK + THREADS_PER_BLOCK * OUTPUT_NODES + OUTPUT_NODES) * sizeof(float), hip_stream>>>(
                d_batch_hidden, d_batch_output_delta, d_weight2, d_batch_hidden_delta, 
                HIDDEN_NODES, OUTPUT_NODES, batch_size
            );
            
            CUDA_CHECK(hipEventRecord(kernel_stop, hip_stream), "Record hidden delta stop event");
            CUDA_CHECK(hipEventSynchronize(kernel_stop), "Synchronize hidden delta stop event");
            CUDA_CHECK(hipEventElapsedTime(&kernel_time, kernel_start, kernel_stop), "Calculate hidden delta time");
            if (should_log) printf("[TIMING] Hidden delta kernel: %.4f ms\n", kernel_time);
            
            // Check for kernel launch errors
            CUDA_CHECK(hipGetLastError(), "calculate_hidden_delta_kernel launch");
            
            // ────────── Update Weights ──────────
            if (should_log) {
                printf("[FUSED] Updating weights...\n");
            }
            
            // Time input-hidden weight update kernel
            CUDA_CHECK(hipEventRecord(kernel_start, hip_stream), "Record input-hidden weights start event");
            
            // Update input-hidden weights (using vectorized4 version)
            int input_hidden_blocks = (INPUT_NODES * HIDDEN_NODES + THREADS_PER_BLOCK - 1) / THREADS_PER_BLOCK;
            input_hidden_blocks = input_hidden_blocks > MAX_BLOCKS ? MAX_BLOCKS : input_hidden_blocks;
            
            update_input_hidden_weights_kernel_vectorized4<<<input_hidden_blocks, THREADS_PER_BLOCK, 
                HIDDEN_NODES * sizeof(float), hip_stream>>>(
                d_batch_input, d_batch_hidden_delta, d_weight1, d_bias1,
                INPUT_NODES, HIDDEN_NODES, batch_size, learning_rate
            );
            
            CUDA_CHECK(hipEventRecord(kernel_stop, hip_stream), "Record input-hidden weights stop event");
            CUDA_CHECK(hipEventSynchronize(kernel_stop), "Synchronize input-hidden weights stop event");
            CUDA_CHECK(hipEventElapsedTime(&kernel_time, kernel_start, kernel_stop), "Calculate input-hidden weights time");
            if (should_log) printf("[TIMING] Input-hidden weights kernel: %.4f ms\n", kernel_time);
            
            // Check for kernel launch errors
            CUDA_CHECK(hipGetLastError(), "update_input_hidden_weights_kernel_vectorized4 launch");
            
            // Time hidden-output weight update kernel
            CUDA_CHECK(hipEventRecord(kernel_start, hip_stream), "Record hidden-output weights start event");
            
            // Update hidden-output weights
            int hidden_output_blocks = (HIDDEN_NODES * OUTPUT_NODES + THREADS_PER_BLOCK - 1) / THREADS_PER_BLOCK;
            hidden_output_blocks = hidden_output_blocks > MAX_BLOCKS ? MAX_BLOCKS : hidden_output_blocks;
            
            update_hidden_output_weights_kernel<<<hidden_output_blocks, THREADS_PER_BLOCK, 
                (THREADS_PER_BLOCK + OUTPUT_NODES) * sizeof(float), hip_stream>>>(
                d_batch_hidden, d_batch_output_delta, d_weight2, d_bias2,
                HIDDEN_NODES, OUTPUT_NODES, batch_size, learning_rate
            );
            
            CUDA_CHECK(hipEventRecord(kernel_stop, hip_stream), "Record hidden-output weights stop event");
            CUDA_CHECK(hipEventSynchronize(kernel_stop), "Synchronize hidden-output weights stop event");
            CUDA_CHECK(hipEventElapsedTime(&kernel_time, kernel_start, kernel_stop), "Calculate hidden-output weights time");
            if (should_log) printf("[TIMING] Hidden-output weights kernel: %.4f ms\n", kernel_time);
            
            // Check for kernel launch errors
            CUDA_CHECK(hipGetLastError(), "update_hidden_output_weights_kernel launch");
        }
        
        // Synchronize to ensure all operations are complete
        CUDA_CHECK(hipStreamSynchronize(hip_stream), "Stream synchronize after all kernels");
        
        // Cleanup timing events
        hipEventDestroy(kernel_start);
        hipEventDestroy(kernel_stop);
    }
    
    // Copy correct count back to host
    CUDA_CHECK(hipMemcpy(&host_correct_count, d_correct_count, sizeof(int), hipMemcpyDeviceToHost), 
               "hipMemcpy for correct count");
    
    // Update the external correct_predictions counter
    *correct_predictions += host_correct_count;
    
    // ────────── Copy Updated Weights Back to Host ──────────
    if (should_log && learning_rate > 0.0f) {
        printf("[FUSED] Copying updated weights back to host...\n");
    }
    
    // Only copy updated weights and biases back if we're training
    if (learning_rate > 0.0f) {
        // Copy updated weights and biases back to host
        CUDA_CHECK(hipMemcpy(weight1, d_weight1, weight1_size, hipMemcpyDeviceToHost), "hipMemcpy for weight1 back");
        CUDA_CHECK(hipMemcpy(weight2, d_weight2, weight2_size, hipMemcpyDeviceToHost), "hipMemcpy for weight2 back");
        CUDA_CHECK(hipMemcpy(bias1, d_bias1, bias1_size, hipMemcpyDeviceToHost), "hipMemcpy for bias1 back");
        CUDA_CHECK(hipMemcpy(bias2, d_bias2, bias2_size, hipMemcpyDeviceToHost), "hipMemcpy for bias2 back");
    }
    
    // ────────── Log Performance and Increment Call Counter ──────────
    if (should_log) {
        end_time = clock();
        double elapsed = (double)(end_time - start_time) / CLOCKS_PER_SEC;
        total_time += elapsed;
        
        printf("[FUSED] Batch training completed in %.6f seconds\n", elapsed);
        printf("[FUSED] Average time per batch: %.6f seconds\n", total_time / (call_count + 1));
        printf("[FUSED] Accuracy for this batch: %.2f%% (%d/%d correct)\n", 
               100.0 * host_correct_count / batch_size, host_correct_count, batch_size);
    }
    
    call_count++;
    return;
    
Error:
    fprintf(stderr, "[FUSED] Error in train_batch_fused, call #%d\n", call_count);
    return;
} 